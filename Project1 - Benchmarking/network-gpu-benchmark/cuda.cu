#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <stdlib.h>
#include <string.h>

double finalGval;
struct timeval t;

__global__ void DoubleAdd(double *d_a,int threadCount,int loopcount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < threadCount)
	{
		for (int j = 0; j < loopcount; ++j)
		{
                        d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = 0.0000001 + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = 0.0000002 + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = 0.0000003 + 0.0000001;
						d_a[i] = 0.0000004 + 0.0000001;
						d_a[i] = 0.0000001 + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = 0.0000002 + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						d_a[i] = 0.0000005 + 0.0000001;
						d_a[i] = 0.0000001 + 0.0000001;
						d_a[i] = d_a[i] + 0.0000001;
						
		}
	}
}

__global__ void IntRunner(int *i_a,int threadCount,int loopcount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < threadCount)
	{
		for (int j = 0; j < loopcount; ++j)
		{
                        i_a[i] = i_a[i] + 41000;
						i_a[i] = 41000 + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 41000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 3 + 41000;
						i_a[i] = 4 + 41000;
						i_a[i] = 1 + 41000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 41000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 5 + 41000;
						i_a[i] = 1 + 41000;
						i_a[i] = i_a[i] + 1;
						
		}
	}
}

__global__ void ShortRunner(short *i_a,int threadCount,int loopcount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < threadCount)
	{
		for (int j = 0; j < loopcount; ++j)
		{
                        i_a[i] = i_a[i] + 30000;
						i_a[i] = 1 + 30000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 30000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 3 + 30000;
						i_a[i] = 4 + 30000;
						i_a[i] = 1 + 30000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 30000;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 5 + 30000;
						i_a[i] = 1 + 30000;
						i_a[i] = i_a[i] + 1;
						
		}
	}
}

__global__ void CharRunner(char *i_a,int threadCount,int loopcount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < threadCount)
	{
		for (int j = 0; j < loopcount; ++j)
		{
                        i_a[i] = i_a[i] + 1;
						i_a[i] = 1 + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 3 + 1;
						i_a[i] = 4 + 1;
						i_a[i] = 1 + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 2 + 1;
						i_a[i] = i_a[i] + 1;
						i_a[i] = 5 + 1;
						i_a[i] = 1 + 1;
						i_a[i] = i_a[i] + 1;
						
		}
	}
}

int main(int argc, char *argv[])
{

	int cudaCores, totalSm, threadCount, loopcount;

			hipSetDevice(0);
			totalSm = 56;
			cudaCores = 3584;
			threadCount = totalSm * cudaCores;
			loopcount = atoi(argv[1]); // loop count
			double *fa, *dfa;
			int *ia, *dia;
			short *sa, *dsa;
			char *ca, *dca;
			double startTime, endTime, curr_time;
			
			//FLOPS
			
			fa = (double *)malloc(threadCount*sizeof(double));
			hipMalloc(&dfa, threadCount*sizeof(double));
			
			for(int i = 0; i < threadCount; ++i)
			{
				fa[i] = 0.00001;
			}
			
			hipMemcpy(dfa, fa, threadCount*sizeof(double), hipMemcpyHostToDevice);
			
			gettimeofday(&t, NULL);
			startTime = t.tv_sec+(t.tv_usec/1000000.0);
			DoubleAdd<<< totalSm, cudaCores >>>(dfa,threadCount,loopcount);
			hipDeviceSynchronize();
			gettimeofday(&t, NULL);
			endTime = t.tv_sec+(t.tv_usec/1000000.0);
			curr_time = endTime - startTime;
			finalGval = (loopcount*totalSm*cudaCores*32*10)/(curr_time*1000000000);
			free(fa);
			hipFree(dfa);
			printf(" FLOPs: %lf G-FLOPs\n", finalGval);
			
			//IOPS 
			
			ia = (int *)malloc(threadCount*sizeof(int));
			hipMalloc(&dia, threadCount*sizeof(int));
			
			for(int i = 0; i < threadCount; ++i)
			{
				ia[i] = 1;
			}
			
			hipMemcpy(dia, ia, threadCount*sizeof(int), hipMemcpyHostToDevice);
			
			gettimeofday(&t, NULL);
			startTime = t.tv_sec+(t.tv_usec/1000000.0);
			IntRunner<<< totalSm, cudaCores >>>(dia,threadCount,loopcount);
			hipDeviceSynchronize();
			gettimeofday(&t, NULL);
			endTime = t.tv_sec+(t.tv_usec/1000000.0);
			curr_time = endTime - startTime;
			finalGval = (loopcount*totalSm*cudaCores*32*10)/(curr_time*1000000000);
			free(ia);
			hipFree(dia);
			printf("IOPS: %lf G-IOPS\n", finalGval);
			
			//HOPS
			
			sa = (short *)malloc(threadCount*sizeof(short));
			hipMalloc(&dsa, threadCount*sizeof(short));
			
			for(int i = 0; i < threadCount; ++i)
			{
				sa[i] = 1;
			}
			
			hipMemcpy(dsa, sa, threadCount*sizeof(short), hipMemcpyHostToDevice);
			
			gettimeofday(&t, NULL);
			startTime = t.tv_sec+(t.tv_usec/1000000.0);
			ShortRunner<<< totalSm, cudaCores >>>(dsa,threadCount,loopcount);
			hipDeviceSynchronize();
			gettimeofday(&t, NULL);
			endTime = t.tv_sec+(t.tv_usec/1000000.0);
			curr_time = endTime - startTime;
			finalGval = (loopcount*totalSm*cudaCores*32*10)/(curr_time*1000000000);
			free(sa);
			hipFree(dsa);
			printf(" HOPS: %lf G-HOPS\n", finalGval);
			
			//QOPS
			
			ca = (char *)malloc(threadCount*sizeof(char));
			hipMalloc(&dca, threadCount*sizeof(char));
			
			for(int i = 0; i < threadCount; ++i)
			{
				ca[i] = 1;
			}
			
			hipMemcpy(dca, ca, threadCount*sizeof(char), hipMemcpyHostToDevice);
			
			gettimeofday(&t, NULL);
			startTime = t.tv_sec+(t.tv_usec/1000000.0);
			CharRunner<<< totalSm, cudaCores >>>(dca,threadCount,loopcount);
			hipDeviceSynchronize();
			gettimeofday(&t, NULL);
			endTime = t.tv_sec+(t.tv_usec/1000000.0);
			curr_time = endTime - startTime;
			finalGval = (loopcount*totalSm*cudaCores*32*10)/(curr_time*1000000000);
			free(ca);
			hipFree(dca);
			printf("QOPS: %lf G-QOPS\n", finalGval);

			
			
	return 0;
}